#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>


__global__
void saxpy(int n, float a, float *x, float *y)
{
	  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) 
	  {
	    y[i] = a * x[i] + y[i];
	    __syncthreads();
	  }
}

int main(void)
{
	int N = 1<<25;
	printf("%d\n", N);
	float *x, *y, *d_x, *d_y;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	  
	x = (float*)malloc(N*sizeof(float));
	y = (float*)malloc(N*sizeof(float));
	hipMalloc(&d_x, N*sizeof(float)); 
	hipMalloc(&d_y, N*sizeof(float));

	for (int n = 0; n < N; n++) {
	  x[n] = 1.0f;
	  y[n] = 2.0f;
	}

	hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

	int numSMs;
	hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);


	printf("Blocks	Threads	Max error	Elapsed time 	Eff. Memory Throughput	"
			 "Peak Memory Throughput	Eff. Computational Throughput\n");
for(int j =1; j < 10; j++){
	for(int i = 1; i < 10; i++){


		int numThreads = i*32, numBlocks = j *32; 	  
		printf("%d	", numBlocks);
                printf("%d	", numThreads);
   		hipEventRecord(start);
		saxpy<<<numBlocks, numThreads>>>(N, 2.0f, d_x, d_y);
		hipEventRecord(stop);

		hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);

		float maxError = 0.0f;
		for (int k = 0; k < N; k++)
		  maxError = fmaxf(maxError, fabsf(y[k]-4.0f));
		printf("%f	", maxError);

		printf("%f	", milliseconds);

		// for Memory Throughput Benchmarking:
		printf("%f GB/s		", 4.0*N*3.0/milliseconds/1.0e6);
		int clockRate;
		int busWidth;
		hipDeviceGetAttribute(&clockRate, hipDeviceAttributeMemoryClockRate, 0);
		hipDeviceGetAttribute(&busWidth, hipDeviceAttributeMemoryBusWidth, 0);
		printf("%f GB/s		", 2.0*clockRate*(busWidth/8.0)/1.0e6);

		// for Computational Throughput Benchmarking:
		// (1* multiply + 1*add) times the amount of vector entries
		printf("%f GFLOP/s	\n",(2*N/milliseconds)/1.0e6);
		}
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(d_x);
	hipFree(d_y);
	free(x);
	free(y);
}
